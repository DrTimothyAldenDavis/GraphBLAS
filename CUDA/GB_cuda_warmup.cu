//------------------------------------------------------------------------------
// GB_cuda_warmup.cu: warmup the GPU
//------------------------------------------------------------------------------

// SPDX-License-Identifier: Apache-2.0
// SuiteSparse:GraphBLAS, Timothy A. Davis, (c) 2017-2019, All Rights Reserved.
// http://suitesparse.com   See GraphBLAS/Doc/License.txt for license.

//------------------------------------------------------------------------------

#include "GB_cuda.h"
/*
#include "rmm/include/rmm/mr/device/managed_memory_resource.hpp"
#include "rmm/include/rmm/mr/device/pool_memory_resource.hpp"
#include "rmm/include/rmm/mr/device/owning_wrapper.hpp"
#include "rmm/include/rmm/mr/device/default_memory_resource.hpp"
#include "rmm/include/rmm/mr/device/per_device_resource.hpp"
#include "rmm/include/rmm/mr/device/cnmem_managed_memory_resource.hpp"
*/
// #include "rmm/detail/cnmem.h"

bool GB_cuda_warmup (int device)
{
    // allocate 'nothing' just to load the drivers.
    // No need to free the result.
    bool ok = GB_cuda_set_device( device );
    if (!ok)
    {
        printf ("invalid GPU: %d\n", device) ;
        return (false) ;
    }

    double gpu_memory_size = GB_Global_gpu_memorysize_get (device);

    printf ("warming up device %d memsize %g sms %d\n",
        device,
        gpu_memory_size, 
        GB_Global_gpu_sm_get (device)) ;

    size_t size = 0 ;
    void *p = GB_malloc_memory (1, 1, &size) ;
    if (p == NULL)
    {
        printf ("Hey!! where's da memory???\n") ;
        return (false) ;
    }
    printf ("oooo nice block of memory of size %lu\n", size) ;
    GB_free_memory (&p, size) ;

    p = hipMalloc (1) ;
    if (p == NULL)
    {
        printf ("Hey!! where's da GPU???\n") ;
        return (false) ;
    }
    hipFree (p) ;

    printf ("GPU %d nice and toasty now\n", device) ;

    // TODO check for jit cache? or in GB_init?

    return  true; //(err == hipSuccess) ;
}

